#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdio>
#include <feval_gpu_xx.h>

// __global__ means this is called from the CPU, and runs on the GPU
__global__ void matrixMul(const double *a, double *b, uint64_t n_col, uint64_t len) {
    // Compute each thread's global row and column index
    uint64_t row = blockIdx.y * blockDim.y + threadIdx.y;
    uint64_t col = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate over row, and down column
    if (row < n_col && col < n_col) {
        double val = 0;
        for (uint64_t k = 0; k < len; k++) {
            val += a[row * len + k] * a[col * len + k];
        }
        b[row * n_col + col] = val;
    }
}

FevalGpuXX::~FevalGpuXX() {
    release();
}

void FevalGpuXX::release() {
    if (d_datum) {
        hipError_t et = hipFree(d_datum);
        if (et == hipSuccess) {
            d_datum = nullptr;
        } else {
            printf("FevalGpuXX::release hipFree d_datum failed %d\n", et);
        }
    }
    if (d_XTX) {
       hipError_t et = hipFree(d_XTX);
        if (et == hipSuccess) {
            d_XTX = nullptr;
        } else {
            printf("FevalGpuXX::release hipFree d_XTX failed %d\n", et);
        } 
    }
}

bool FevalGpuXX::init(uint64_t _n_col, uint64_t _len) {
    n_col = _n_col;
    uint32_t THREADS = 32;
    BLOCKS = (n_col + THREADS - 1) / THREADS;
    hipMalloc(&d_XTX, n_col * n_col * sizeof(double));
    if (d_XTX == nullptr) return false;
    hipError_t et = hipMalloc(&d_datum, _len * sizeof(double));
    if (et == hipSuccess) return true;
    else {
        printf("FevalGpuXX::init hipMalloc failed ec=%d, %p,%zu,%zu\n", et, d_datum, n_col, _len);
        return false;
    }
}

void FevalGpuXX::calc(const std::vector<std::vector<double>*>& pXs, double * h_XTX, uint64_t offset, uint64_t _len) {
    uint32_t THREADS = 32;
    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);
    for (size_t i = 0; i < n_col; i++) {
        hipMemcpy(d_datum + i * _len, pXs[i]->data() + offset, _len * sizeof(double), hipMemcpyHostToDevice);
    }
    matrixMul<<<blocks, threads>>>(d_datum, d_XTX, n_col, _len);
    hipMemcpy(h_XTX, d_XTX, n_col * n_col * sizeof(double), hipMemcpyDeviceToHost);
    // printf("FevalGpuXX::calc hipMemcpyDeviceToHost %f %zu,%zu\n", h_XTX[0], n_col, new_len);
}